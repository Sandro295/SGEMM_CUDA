#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void kernel(unsigned int *A, unsigned int *B, int row) {
  auto x = threadIdx.x / 4;
  auto y = threadIdx.x % 4;
  A[x * row + y] = x;
  B[x * row + y] = y;
}

int main(int argc, char **argv) {
  unsigned int *Xs, *Ys;
  unsigned int *Xs_d, *Ys_d;

  unsigned int SIZE = 4;

  Xs = (unsigned int *)malloc(SIZE * SIZE * sizeof(unsigned int));
  Ys = (unsigned int *)malloc(SIZE * SIZE * sizeof(unsigned int));

  hipMalloc((void **)&Xs_d, SIZE * SIZE * sizeof(unsigned int));
  hipMalloc((void **)&Ys_d, SIZE * SIZE * sizeof(unsigned int));

  dim3 grid_size(1, 1, 1);
  dim3 block_size(4 * 4);

  kernel<<<grid_size, block_size>>>(Xs_d, Ys_d, 4);

  hipMemcpy(Xs, Xs_d, SIZE * SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipMemcpy(Ys, Ys_d, SIZE * SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  for (int row = 0; row < SIZE; ++row) {
    for (int col = 0; col < SIZE; ++col) {
      std::cout << "[" << Xs[row * SIZE + col] << "|" << Ys[row * SIZE + col]
                << "] ";
    }
    std::cout << "\n";
  }

  hipFree(Xs_d);
  hipFree(Ys_d);
  free(Xs);
  free(Ys);
}
